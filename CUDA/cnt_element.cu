
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <random>
#include <vector>
#include <chrono>
#include <deque>
#include <algorithm>
#include <iterator>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 1024

__global__ void cnt_reduce(int *arr, const int n, const int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < m) {
        int j = n-i-1;
        if (i != j) {
            arr[i] = arr[i] + arr[j];
        }
    }
}

int count_size(int *cnt_arr, int n) {
    int m = (n+1)/2;
    
    while (n > 1) {
        cnt_reduce<<<(m + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(cnt_arr, n, m);
        n = m;
        m = (n+1)/2;
    }
    hipDeviceSynchronize();
    return cnt_arr[0];
}

void random_vector(int *arr, const int n, const int min_val=0.0, const int max_val=1000.0) {
    static std::random_device rd;
    static std::mt19937 mte(rd());
    std::uniform_int_distribution<int> dist(min_val, max_val);
    
    for (int i = 0; i < n; i++) {
        arr[i] = dist(mte);
    }
}

bool check_correctness(int *arr, int pred, int n) {
    int cnt = 0;
    for (int i = 0; i < n; i++) {
        cnt += arr[i];
    }
    return pred == cnt;
}

int main(void) {
    int n = 1 << 30;
    
    int *arr, *temp;
    hipMallocManaged(&arr, n*sizeof(int));
    
    random_vector(arr, n, 0, 1);
    
    temp = new int[n];
    std::copy(arr, arr+n, temp);
    
    auto t1 = std::chrono::high_resolution_clock::now();
    int cnt = count_size(arr, n);
    auto t2 = std::chrono::high_resolution_clock::now();
    
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>( t2 - t1 ).count();

    std::cout << duration << std::endl;
    
    t1 = std::chrono::high_resolution_clock::now();
    std::cout << check_correctness(temp, cnt, n) << std::endl;
    t2 = std::chrono::high_resolution_clock::now();
    
    duration = std::chrono::duration_cast<std::chrono::milliseconds>( t2 - t1 ).count();

    std::cout << duration << std::endl;
    
    hipFree(arr);

    return 0;
}
