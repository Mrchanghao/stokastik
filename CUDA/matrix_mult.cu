
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <random>
#include <vector>
#include <chrono>

#define TILE_DIM 32
#define M_DIM 50

__global__ void multiplyNaive(const int *mat_1, const int *mat_2, int *mat_prod, const int n, const int p) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < n && x < p) {
        int sum = 0;
    
        for (int i = 0; i < M_DIM; i++) {
            sum += mat_1[y*M_DIM+i] * mat_2[i*p+x];
        }
    
        mat_prod[y*p+x] = sum;
    }
}

__global__ void multiplySharedMem(const int *mat_1, const int *mat_2, int *mat_prod, const int n, const int p) {
    __shared__ float aTile[TILE_DIM][M_DIM], bTile[M_DIM][TILE_DIM];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    int h = (M_DIM + blockDim.x - 1)/blockDim.x;
    
    int start_a = h*threadIdx.x;
    int end_a = M_DIM < h*(threadIdx.x+1)?M_DIM:h*(threadIdx.x+1);
    
    for (int i = start_a; i < end_a; i++) {
        aTile[threadIdx.y][i] = mat_1[y*M_DIM+i];
    }
    
    h = (M_DIM + blockDim.y - 1)/blockDim.y;
    
    int start_b = h*threadIdx.y;
    int end_b = M_DIM < h*(threadIdx.y+1)?M_DIM:h*(threadIdx.y+1);
    
    for (int i = start_b; i < end_b; i++) {
        bTile[i][threadIdx.x] = mat_2[i*p+x];
    }
    
    __syncthreads();
    
    if (y < n && x < p) {
        int sum = 0;
    
        for (int i = 0; i < M_DIM; i++) {
            sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
        }
    
        mat_prod[y*p+x] = sum;
    }
}

std::vector<std::vector<int>> random_matrix(const int num_rows, const int num_cols, const int min_val=0.0, const int max_val=1000.0) {
    std::vector<std::vector<int>> my_arr;
    static std::random_device rd;
    static std::mt19937 mte(rd());
    std::uniform_int_distribution<int> dist(min_val, max_val);
    
    for (int i = 0; i < num_rows; i++) {
        std::vector<int> my_arr_col;
        for (int j = 0; j < num_cols; j++) {
            my_arr_col.push_back(dist(mte));
        }
        my_arr.push_back(my_arr_col);
    }
    
    return my_arr;
}

bool check_correctness(const int *mat_1, const int *mat_2, int *mat_prod, const int n, const int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            int sum = 0;
            for (int k = 0; k < M_DIM; k++) {
                sum += mat_1[i*M_DIM+k] * mat_2[k*p+j];
            }
            if (sum != mat_prod[i*p+j]) {
                return false;
            }
        }
    }
    return true;
}

int main(void) {
    int n = 5000;
    int p = 8000;

    dim3 dimGrid((p + TILE_DIM - 1)/TILE_DIM, (n + TILE_DIM - 1)/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
    
    int *mat_1, *mat_2, *mat_prod; 

    hipMallocManaged(&mat_1, n*M_DIM*sizeof(int));
    hipMallocManaged(&mat_2, M_DIM*p*sizeof(int));
    hipMallocManaged(&mat_prod, n*p*sizeof(int));
    
    std::vector<std::vector<int>> my_arr_1 = random_matrix(n, M_DIM, 0, 10);
    std::vector<std::vector<int>> my_arr_2 = random_matrix(M_DIM, p, 0, 10);
    
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < M_DIM; j++) {
            mat_1[M_DIM*i + j] = my_arr_1[i][j];
        }
    }
    
    for (int i = 0; i < M_DIM; i++) {
        for (int j = 0; j < p; j++) {
            mat_2[p*i + j] = my_arr_2[i][j];
        }
    }
    
    auto t1 = std::chrono::high_resolution_clock::now();
    multiplySharedMem<<<dimGrid, dimBlock>>>(mat_1, mat_2, mat_prod, n, p);
    hipDeviceSynchronize();
    auto t2 = std::chrono::high_resolution_clock::now();
    
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>( t2 - t1 ).count();

    std::cout << duration << std::endl;
    std::cout << check_correctness(mat_1, mat_2, mat_prod, n, p) << std::endl;
    
    return 0;
}

